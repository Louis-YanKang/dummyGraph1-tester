
#include <hip/hip_runtime.h>
#include <iostream>

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600


__device__ int myAtomicAdd(int *address, int incr)
{
    // Create an initial guess for the value stored at *address.
    int guess = *address;
    int oldValue = atomicCAS(address, guess, guess + incr);

    // Loop while the guess is incorrect.
    while (oldValue != guess)
    {
        guess = oldValue;
        oldValue = atomicCAS(address, guess, guess + incr);
    }

    return oldValue;
}

template<typename T>
__device__ T fetch_and_add(T *x, T inc) {
    T orig_val = myAtomicAdd(x,inc);
    return orig_val;
}

__global__ void kernel(int *sharedInteger)
{
    myAtomicAdd(sharedInteger, 1);
}

//template __device__ int fetch_and_add<int>(int*, int);

#endif
