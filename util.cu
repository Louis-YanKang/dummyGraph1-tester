
#include <hip/hip_runtime.h>
#include <iostream>

#if __CUDA_ARCH__ < 600
__device__ int myAtomicAdd(int *address, int incr)
{
    // Create an initial guess for the value stored at *address.
    int guess = *address;
    int oldValue = atomicCAS(address, guess, guess + incr);

    // Loop while the guess is incorrect.
    while (oldValue != guess)
    {
        guess = oldValue;
        oldValue = atomicCAS(address, guess, guess + incr);
    }

    return oldValue;
}
#endif

__global__ void kernel(int *sharedInteger)
{
    myAtomicAdd(sharedInteger, 1);
}

