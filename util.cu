#include "hip/hip_runtime.h"
#include <iostream>
#include "dummyGraph1.h"


__device__ int myAtomicAdd(int *address, int incr)
{
    // Create an initial guess for the value stored at *address.
    int guess = *address;
    int oldValue = atomicCAS(address, guess, guess + incr);

    // Loop while the guess is incorrect.
    while (oldValue != guess)
    {
        guess = oldValue;
        oldValue = atomicCAS(address, guess, guess + incr);
    }

    return oldValue;
}


__global__ void kernel(int *sharedInteger)
{
    myAtomicAdd(sharedInteger, 1);
}

